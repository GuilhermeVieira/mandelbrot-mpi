#include "hip/hip_runtime.h"
#include <iostream>
#include "mandel.cuh"
#define INTER_LIMIT 255

__device__ int get_inter (thrust::complex<float> c) {
    int i;
    thrust::complex<float> z(0.0, 0.0);

    for (i = 0; i < INTER_LIMIT; ++i) {
        if (thrust::abs(z) > 2 ) {
            break;
        }
        z = thrust::pow(z, 2) + c;
    }
    return i;
}

__global__ void fill_matrix (int *res, const int start, const int w, const int work_size, thrust::complex<float> c0, const float del_x, const float del_y, const int threads, const int blocks, const int offset) {
    thrust::complex<float> del(0, 0);
    unsigned int k = threadIdx.x + blockIdx.x*threads + blocks*threads*offset;
    if (k >= work_size)
        return;
    del.real(del_x * ((start + k) % w));
    del.imag(del_y * ((start + k) / w));
    res[k] = get_inter(c0 + del);
    return;
}

__host__ void prepare (int *res_matrix, const int start, const int w, const int work_size, thrust::complex<float> c0, const float del_x, const float del_y, const int threads) {
    int *d_res_matrix; 
    int *d_w; 
    int *d_start;
    int *d_work_size;
    float *d_del_x;
    float *d_del_y;
    thrust::complex<float> *d_c0;  
    
    hipSetDevice(0);

    if (hipSuccess != hipMallocManaged((void **) &d_res_matrix, sizeof(int)*work_size)) {
        std::cerr << "Could not allocate memory";
        exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMallocManaged((void **) &d_start, sizeof(int))) {
    	std::cerr << "Could not allocate memory";
	exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMallocManaged((void **) &d_w, sizeof(int))) {
        std::cerr << "Could not allocate memory";
        exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMallocManaged((void **) &d_work_size, sizeof(int))) {
    	std::cerr << "Could not allocate memory";
	exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMallocManaged((void **) &d_c0, sizeof(thrust::complex<float>)) ) {
        std::cerr << "Could not allocate memory";
        exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMallocManaged((void **) &d_del_y, sizeof(float))) {
        std::cerr << "Could not allocate memory";
        exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMallocManaged((void **) &d_del_x, sizeof(float))) {
        std::cerr << "Could not allocate memory";
        exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMemcpy(d_start, &start, sizeof(int), hipMemcpyHostToDevice)) {
    	std::cerr << "Could not copy memory";
	exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMemcpy(d_w, &w, sizeof(int), hipMemcpyHostToDevice)) {
        std::cerr << "Could not copy memory";
        exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMemcpy(d_work_size, &work_size, sizeof(int), hipMemcpyHostToDevice)) {
    	std::cerr << "Could not copy memory";
	exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMemcpy(d_c0, &c0, sizeof(thrust::complex<float>), hipMemcpyHostToDevice)) {
        std::cerr << "Could not copy memory";
        exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMemcpy(d_del_y, &del_y, sizeof(float), hipMemcpyHostToDevice)) {
        std::cerr << "Could not copy memory";
        exit(EXIT_FAILURE);
    }
    if (hipSuccess != hipMemcpy(d_del_x, &del_x, sizeof(float), hipMemcpyHostToDevice)) {
        std::cerr << "Could not copy memory";
        exit(EXIT_FAILURE);
    }
    
    int block = 1024;
    int max = (work_size/ (threads*block)) + 1;
    for (int i = 0; i < max; ++i) {
        fill_matrix<<<block, threads>>> (d_res_matrix, *d_start, *d_w, *d_work_size, *d_c0, *d_del_x, *d_del_y, threads, block, i);
        hipDeviceSynchronize();
    }
    
    if (hipSuccess != hipMemcpy(res_matrix, d_res_matrix, sizeof(int)*work_size, hipMemcpyDeviceToHost)) {
        std::cerr << "Could not copy memory";
        exit(EXIT_FAILURE);
    }
    return;
}
